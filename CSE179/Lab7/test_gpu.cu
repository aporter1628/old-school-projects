#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#define N 2048


__global__ void findMax(int *a, int *b){
        b[0] = 0;
        if(a[threadIdx.x] > b[0]){
                b[0] = a[threadIdx.x];
        }
        __syncthreads();
}

int findMaxCPU(int *a){
        int max = 0;
        for(int i = 0; i < N; i++){
                if(a[i] > max) {
                        max = a[i];
                }
        }
        return max;
}

int main(int argc, char *argv[]){
        int *a, *b;
        int *d_a, *d_b;
        int size = N * sizeof(int);

         struct timeval before;
        struct timeval after;

        hipMalloc((void **)&d_a, size);
        hipMalloc((void**)&d_b, size);
        a = (int *)malloc(size);
        b = (int*)malloc(size);
        //random_ints(a, N);
        for(int i = 0; i < N; i++){
                a[i] = rand();
        }

        hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);


        gettimeofday(&before, NULL);
        findMax<<<1,1>>>(d_a, d_b);

        gettimeofday(&after, NULL);

        hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);
        printf("For GPU, the max is %d and it takes %d microseconds\n", b[0], (after.tv_sec * 1000000 + after.tv_usec) - (before.tv_sec * 1000000 + before.tv_usec));
        gettimeofday(&before, NULL);
        int max = findMaxCPU(a);
        gettimeofday(&after, NULL);

        printf("For CPU, the max is %d and it takes %d microseconds\n", max, (after.tv_sec * 1000000 + after.tv_usec) - (before.tv_sec * 1000000 + before.tv_usec));
        free(a); free(b);
        hipFree(d_a);
        hipFree(d_b);

        return 0;
}